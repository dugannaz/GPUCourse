#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <hipfft/hipfft.h>

#define NX 2048

using namespace std;

int main(int argc, char *argv[]) {

        struct timeval tt1, tt2;
        int ms;
        float fms;

	// create cufft plan
        hipfftHandle plan;
        hipfftPlan2d(&plan, NX,NX, HIPFFT_Z2Z);

  	// allocate hipfftDoubleComplex type host memory 
        hipfftDoubleComplex *data;
	data = (hipfftDoubleComplex*)malloc(NX*NX * sizeof(hipfftDoubleComplex));

	// data initialization
         for(int j=0 ; j < NX ; j++)
           for(int k=0 ; k < NX ; k++) {
                data[k + j*NX].x = sin(double(j)+double(k));
                data[k + j*NX].y = cos(double(j)+double(k));
           }

	// check initial value of a data element 
        cout << "initial value = " << data[43].x << " + " 
		<< data[43].y << "i" << endl;

	// allocate hipfftDoubleComplex type device memory
	hipfftDoubleComplex *devPtr;
        hipMalloc((void**)&devPtr, sizeof(hipfftDoubleComplex)*NX*NX);

	// copy data to device memory
        hipMemcpy(devPtr, data, sizeof(hipfftDoubleComplex)*NX*NX, hipMemcpyHostToDevice);

        hipDeviceSynchronize();
        gettimeofday( &tt1, NULL );

        // run fft 
      
        hipfftExecZ2Z(plan, devPtr, devPtr, HIPFFT_FORWARD);	

        hipDeviceSynchronize();
        gettimeofday( &tt2, NULL );

        // make inverse transform 
        
        hipfftExecZ2Z(plan, devPtr, devPtr, HIPFFT_BACKWARD);	

	// transfer result back from device
        hipMemcpy(data, devPtr, sizeof(hipfftDoubleComplex)*NX*NX, hipMemcpyDeviceToHost);

        // destroy cufft plan
        hipfftDestroy(plan);

	// free device memory
        hipFree(devPtr);

	// check initial value of the same data element. Initial and final values should match
	// after a forward and inverse transform. 
        cout << "final value   = " << data[43].x/double(NX*NX) << " + " 
		<< data[43].y/double(NX*NX) << "i" << endl;

	// free host memory
	free(data);

	// timing
        ms = (tt2.tv_sec - tt1.tv_sec);
        ms = ms * 1000000 + (tt2.tv_usec - tt1.tv_usec);
        fms = ((double)ms)/1000000.f;
        cout << "Computation time = " << fms << " seconds" << endl;

}

