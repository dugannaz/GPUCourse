#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>
#include <stdio.h>

using namespace std;

/* Overlapping data transfers and kernel execution
 * - pinned memory
 * - streams
 * - different strategies depending on concurrent data transfers enabled or not
 */

#define TILE_DIM    16
#define BLOCK_ROWS  16

__global__ void transposeNaive(double *odata, double* idata, int width, int height, int nreps)
{
  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

  int index_in  = xIndex + width * yIndex;
  int index_out = yIndex + height * xIndex;
  for (int r=0; r < nreps; r++) {
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      odata[index_out+i] = idata[index_in+i*width];
    }
  }
}

int main() {


	// check if device can overlap data transfers with computation
	int deviceCount; 
	hipGetDeviceCount(&deviceCount); 
	int device; 
	for (device = 0; device < deviceCount; ++device) { 
		hipDeviceProp_t deviceProp; 
		hipGetDeviceProperties(&deviceProp, device); 
		printf("Device %d has compute capability %d.%d.\n", device, 
			deviceProp.major, deviceProp.minor);
		cout << " asyncEngineCount = " << deviceProp.deviceOverlap << endl;
	}
	/************************/

	// side length of square matrix
        int side = 2048;

        // number of elements in a single matrx
        int n = side*side;

        // number of matrices to transpose
        int nTranspose = 96;

        // number of transpose operations on a single matrix
        int nreps = 20;


	int nStream = 8;
	// (*) define streams here
	
      	hipStream_t stream[nStream];
        for (int i=0; i<nStream; i++)
                hipStreamCreate(&stream[i]);

	// create events
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

	// (*) modify here to allocate pinned host memory with hipHostMalloc 
	double *data;
	hipHostMalloc((void**) &data, nTranspose * n * sizeof(double));
     
	// data initialization
	for (int j=0; j<nTranspose; j++)	   
	for (int i=0; i<n; i++) {
                data[i+j*n] = double(i+j*n);
        }

	double *data_dev;
	// (*) modify device memory allocation size according to nStream
        hipMalloc((void**) &data_dev, nStream * 2 * n * sizeof(double));

	dim3 grid(side/16,side/16,1);
        dim3 threads(16,16,1);

	// record start event
        hipEventRecord(start, 0);

	// concurrent data transfers not supported
      for (int j=0; j<nTranspose/nStream; j++) {
        for (int i=0; i<nStream; i++) {
        
		int ij = j*nStream+i;        
		int offset = i * n;

		hipMemcpyAsync(data_dev + offset*2, data + ij*n, n * sizeof(double), 
					hipMemcpyHostToDevice, stream[i]);
                
	}

        for (int i=0; i<nStream; i++) {

		int offset = i * n;

        	transposeNaive <<< grid, threads, 0, stream[i]  >>>
				(data_dev + offset*2+n, data_dev + offset*2, side, side, nreps);
	}
	
        for (int i=0; i<nStream; i++) {

		int ij = j*nStream+i;
		int offset = i * n;

        	hipMemcpyAsync(data + ij*n, data_dev + offset*2 + n, n * sizeof(double), 
					hipMemcpyDeviceToHost, stream[i]);
        }
      }

	// record stop event
        hipEventRecord(stop, stream[nStream-1]);

	// elapsed time
        hipEventSynchronize(stop);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);
        cout << "Comp time = " << elapsedTime/1000.0 << endl;

        // destroy events
        hipEventDestroy(start);
        hipEventDestroy(stop);
	

	// (*) Destroy streams here
	for (int i=0; i<nStream; i++)
                hipStreamDestroy(stream[i]);


	hipFree(data_dev);

	cout << "value check = " << data[n+5467] << endl;
	
	// (*) modify here to free pinned host memory
	hipHostFree(data);
}

