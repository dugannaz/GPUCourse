#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <hipfft/hipfft.h>

/* hipfftPlanMany function
 * - 2D FFT transform using 1D cuffts without transpositions
 * - compare with transposition version 
 */

#define NX 512
#define BATCH NX
#define TILE_DIM  16

using namespace std;

__global__ void transposeNoBankConflicts(hipfftDoubleComplex *idata, hipfftDoubleComplex *odata, int width, int height)
{
  __shared__ float tile[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

      tile[threadIdx.y][threadIdx.x] = idata[index_in].x;

    __syncthreads();

      odata[index_out].x = tile[threadIdx.x][threadIdx.y];

    // (*) transpose also the complex part of the matrix
    __syncthreads();

      tile[threadIdx.y][threadIdx.x] = idata[index_in].y;

    __syncthreads();

      odata[index_out].y = tile[threadIdx.x][threadIdx.y];


}

int main(int argc, char *argv[]) {

        struct timeval tt1, tt2;
        int ms;
        float fms;

	// create cufft plan
        hipfftHandle plan;
        hipfftPlan1d(&plan, NX, HIPFFT_Z2Z, BATCH);

  	// allocate hipfftDoubleComplex type host memory 
        hipfftDoubleComplex *data;
	data = (hipfftDoubleComplex*)malloc(NX*BATCH * sizeof(hipfftDoubleComplex));

	// data initialization
         for(int j=0 ; j < BATCH ; j++)
           for(int k=0 ; k < NX ; k++) {
                data[k + j*NX].x = sin(double(j)+double(k));
                data[k + j*NX].y = cos(double(j)+double(k));
           }

	// check initial value of a data element 
        cout << "initial value = " << data[43].x << " + " 
		<< data[43].y << "i" << endl;

	// allocate hipfftDoubleComplex type device memory
	hipfftDoubleComplex *devPtr;
        hipMalloc((void**)&devPtr, sizeof(hipfftDoubleComplex)*NX*BATCH*2);

	// copy data to device memory
        hipMemcpy(devPtr, data, sizeof(hipfftDoubleComplex)*NX*BATCH, hipMemcpyHostToDevice);

        // create plans
        hipfftHandle plan1;
        hipfftHandle plan2;

        int inembed[1];
        int onembed[1];

        inembed[0] = 1;
        onembed[0] = 1;

	// (*) define FFT dimension array
        int n1d[3]= {NX, 1, 1};

	// (*) planning for X dimension
 	hipfftPlanMany(&plan1,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, HIPFFT_Z2Z, NX);
 
	// (*) planning for Y dimension
        hipfftPlanMany(&plan2,  1, n1d,
              inembed, NX, 1,
              onembed, NX, 1, HIPFFT_Z2Z, NX);


        hipDeviceSynchronize();
        gettimeofday( &tt1, NULL );

        
        // X transform 
	hipfftExecZ2Z(plan1, devPtr, devPtr + NX*NX, HIPFFT_FORWARD);

   	//Y transform
     	hipfftExecZ2Z(plan2, devPtr + NX*NX, devPtr, HIPFFT_FORWARD);

        hipDeviceSynchronize();
        gettimeofday( &tt2, NULL );

	// timing
        ms = (tt2.tv_sec - tt1.tv_sec);
        ms = ms * 1000000 + (tt2.tv_usec - tt1.tv_usec);
        fms = ((double)ms)/1000000.f;
        cout << "Computation time = " << fms << " seconds" << endl;

        // runtime configuration parameters for transposition
        dim3 grid(NX/TILE_DIM,NX/TILE_DIM,1);
        dim3 threads(TILE_DIM,TILE_DIM,1);

        // (*) make inverse transform 
        for (int tt=0; tt<2; tt++) {
                hipfftExecZ2Z(plan, devPtr, devPtr + NX*NX, HIPFFT_BACKWARD);
		transposeNoBankConflicts <<< grid, threads >>>(devPtr + NX*NX, devPtr, NX, NX);
        }

        hipDeviceSynchronize();
        gettimeofday( &tt1, NULL );

	// transfer result back from device
        hipMemcpy(data, devPtr, sizeof(hipfftDoubleComplex)*NX*BATCH, hipMemcpyDeviceToHost);

        // (*) destroy cufft plan
        hipfftDestroy(plan);

	// free device memory
        hipFree(devPtr);

	// check initial value of the same data element. Initial and final values should match
	// after a forward and inverse transform. 
        cout << "final value   = " << data[43].x/double(NX*NX) << " + " 
		<< data[43].y/double(NX*NX) << "i" << endl;

	// free host memory
	free(data);

	// timing
        ms = (tt1.tv_sec - tt2.tv_sec);
        ms = ms * 1000000 + (tt1.tv_usec - tt2.tv_usec);
        fms = ((double)ms)/1000000.f;
        cout << "Computation time = " << fms << " seconds" << endl;

}

