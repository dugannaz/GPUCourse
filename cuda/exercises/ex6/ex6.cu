#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>

using namespace std;

#define nPerThread 32

/* Simple Cuda Program: Shared memory
 * - Use dynamic shared memory
 * - bank conflicts
 * - synchronization
 */
 

// no bank conflicts
__global__ void addOneShared(const int n, double *data) {

	// (*) define dynamic shared memory

	int nt = blockDim.x;
	int t = threadIdx.x;
	int b = blockIdx.x;	

	// (*) copy data of the corresponding block to shared memory

	// (*) make addition in the shared memory

	// (*) copy data back to global memory
	
}

// bank conflicts
__global__ void addOneShared_bankConflits(const int n, double *data) {

	// (*) define dynamic shared memory

	int nt = blockDim.x;
	int t = threadIdx.x;
	int b = blockIdx.x;	

	// (*) copy data of the corresponding block to shared memory

	// (*) make addition in the shared memory

	// (*) copy data back to global memory
	
}

int main() {

	time_t sTime = time(NULL);
        struct timeval tt1, tt2;
        int ms;
        double fms;

	int nBlocks = 256;
	int nThreads = 128;

	int n = nPerThread*nThreads*nBlocks;

	double *data = (double*) malloc(n * sizeof(double));
        for (int i=0; i<n; i++) {
                data[i] = i;
        }

	double *data_dev;
        hipMalloc((void**) &data_dev, n * sizeof(double));

	hipMemcpy(data_dev, data, n * sizeof(double) , hipMemcpyHostToDevice);
	hipError_t error = hipGetLastError();
	cout << "copy to device = " << error << " : " << hipGetErrorString(error) << endl;

	hipDeviceSynchronize();
        gettimeofday( &tt1, NULL );

	int sharedMem = nThreads * nPerThread * sizeof(double);

	// (*) Add shared memory size to execution configuration parameters
        addOneShared <<< nBlocks, nThreads >>>(n, data_dev);

	error = hipGetLastError();
        cout << "run kernel = " << error << " : " << hipGetErrorString(error) << endl;

	hipDeviceSynchronize();
        gettimeofday( &tt2, NULL );

	ms = (tt2.tv_sec - tt1.tv_sec);
        ms = ms * 1000000 + (tt2.tv_usec - tt1.tv_usec);
        fms = ((double)ms)/1000000.0;
        cout << "Comp time = " << fms << endl;

        hipMemcpy(data, data_dev, n * sizeof(double) , hipMemcpyDeviceToHost);
	error = hipGetLastError();
        cout << "copy from device = " << error << " : " << hipGetErrorString(error) << endl;

	hipFree(data_dev);
	

	cout << "data[n-1] = " << data[n-1] << endl;
	free(data);
}



