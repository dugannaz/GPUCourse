#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <sys/time.h>

using namespace std;

/* Reduction
 * - reduction sum in shared memory
 */

__global__ void reduction(double *data, double *result) {

	int gid = threadIdx.x + blockIdx.x * blockDim.x *2;
	int id = threadIdx.x;

	// ( bitwise operator s >>= n : s = s / (2^n) )

	for(int s=blockDim.x; s>0; s>>=1) { 
        	if (id < s) {
            		data[gid] += data[gid + s];
        	}
        	__syncthreads();
    	}

	if (id == 0) 
		result[blockIdx.x] = data[gid];
}

__global__ void reductionShared(double *data, double *result) {

	// (*) define dynamic shared memory
	

	int gid = threadIdx.x + blockIdx.x * blockDim.x *2;
	int id = threadIdx.x;

	// (*) copy block data to shared memory
	

	// (*) synchronize threads
	

	// (*) do reduction in shared memory
	

	// (*) copy block result to result array
	

}

int main(int argc, char *argv[]) {

	// time variables
	time_t sTime = time(NULL);
        struct timeval tt1, tt2;
        int ms;
        double fms;

	// data size
	int n = 4194304; // (= 2^22)

	// (*) set block size to its maximum value
	int nThreads = ;
	int nBlocks = n/(2*nThreads);
	
	// data and result arrays
	double *devData, *hostData;
	double *devResult, *hostResult;

	// host memory allocation 
        hostData = (double*)malloc(n*sizeof(double));
	hostResult = (double*)malloc(nBlocks*sizeof(double));

	// data initializtion
	for (int i=0; i<n; i++)
		hostData[i] = (double)i;

	// device memory allocation
        hipMalloc((void**)&devData, n*sizeof(double));
	hipMalloc((void**)&devResult, nBlocks*sizeof(double));

	// copy data to device
	hipMemcpy(devData, hostData, n*sizeof(double), hipMemcpyHostToDevice);	

	// kernel run with timing
	hipDeviceSynchronize();
        gettimeofday( &tt1, NULL );

	// (*) modify kernel run for shared memory version
        reduction <<< nBlocks, nThreads >>> (devData, devResult);

	hipDeviceSynchronize();
        gettimeofday( &tt2, NULL );

	// transfer results from device
        hipMemcpy(hostResult, devResult, nBlocks*sizeof(double), hipMemcpyDeviceToHost);

        // summation of block results
	double total = 0;
	for (int i=0; i<nBlocks; i++) {
		total += hostResult[i];
	}
	
	// screen output of result
	cout << "Total = " << total << endl;

	// time calculation
	ms = (tt2.tv_sec - tt1.tv_sec);
        ms = ms * 1000000 + (tt2.tv_usec - tt1.tv_usec);
        fms = ((double)ms)/1000000.0;
        cout << "Comp time = " << fms << endl;

	// cleanup 
        hipFree(devData);
	hipFree(devResult);
	free(hostData);
	free(hostResult);
}

