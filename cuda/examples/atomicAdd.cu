#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>

using namespace std;

/* example for atomic function usage
 */

__global__ void atomic(int n, float *a) {

	//a[0] += 1.0f; // gives wrong result
	
	// instead use atomic function
	atomicAdd(&a[0], 1.0f); 
}

int main() {

	int n = 1024;

	float *data = (float*) malloc(n * sizeof(float));
        for (int i=0; i<n; i++) {
                data[i] = (float)i;
        }

	float *data_dev;
        hipMalloc((void**) &data_dev, n * sizeof(float));

	hipMemcpy(data_dev, data, n * sizeof(float) , hipMemcpyHostToDevice);
	hipError_t error = hipGetLastError();
	cout << "copy to device = " << error << " : " << hipGetErrorString(error) << endl;

        int nBlocks = 1;
	int nThreads = 1024;

	atomic <<< nBlocks, nThreads  >>>(n, data_dev);

	error = hipGetLastError();
        cout << "run kernel = " << error << " : " << hipGetErrorString(error) << endl;

        hipMemcpy(data, data_dev, n * sizeof(float) , hipMemcpyDeviceToHost);
	error = hipGetLastError();
        cout << "copy from device = " << error << " : " << hipGetErrorString(error) << endl;

	hipFree(data_dev);

	cout << "data[0] = " << data[0] << endl;
	free(data);
}



